
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 1073741824
#define THREADS_PER_BLOCK 512

// TITLE: Vector Addition Threads

// add will be run in parallel, in many threads in a single Thread Block. 
// Each thread knows which thread it is in via threadIdx.x

// y = 1 if e in x[0:n], 0, otherwise
__global__ void find(int n, int e, int *x, int *y){
    int i = threadIdx.x;
    // if (i < 10) printf("%i - %i\n", x[i], e);
    if (i < n && x[i] == e){
        *y = 1;
    }
}

void random_ints(int* a, int n) {
   for (int i = 0; i < n; ++i)a[i] = rand()%1000;
}

int main(void) {

	double size = N*sizeof(int);
    int *a;
    int *y;
    int e = 10;

 	// Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&a, size);
    hipMallocManaged(&y, sizeof(int));
    *y = 0;
    random_ints(a,N);
    a[5] = 10;
	find<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(N, e, a, y);

  	// Wait for GPU to finish before accessing on host
  	hipDeviceSynchronize();

    // for(int i = 0; i < 10; i++){
    //     printf("%i, ", a[i]);
    // }
	printf("Found %i: %i\n", e, *y);

	// Free memory
	hipFree(a); hipFree(y);

	return 0;
}


