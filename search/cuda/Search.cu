
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

int bruteIncrement(char* brute, int alphabetLen, int wordLen, int incrementBy) {
	int i = 0;
	while(incrementBy > 0 && i < wordLen) {
		int add = incrementBy + brute[i];
		brute[i] = (char)(add % alphabetLen);
		incrementBy = add / alphabetLen;
		i++;
	}
	return incrementBy == 0; 
}

__device__ void cudaStrCmp(char *a, char *b, int len, int* res) {
	printf("STRCMP %s %s\n", a, b);
	for (int i = 0; i < len; i++) {
		if (a[i] != b[i]) {
			*res = 0;
			return;
		}
	}
}

__device__ void k_bruteIncrement(char* brute, int alphabetLen, int wordLen, int incrementBy, int *incRes) {
	int i = 0;
	while(incrementBy > 0 && i < wordLen) {
		int add = incrementBy + brute[i];
		brute[i] = (char)(add % alphabetLen);
		incrementBy = add / alphabetLen;
		i++;
	}
	if (incrementBy == 0) {
		*incRes = 1;
	}
}

__device__ void bruteToString(char *brute, int wordLen, char *alphabet, char *out){
	for(int i=0;i<wordLen;i++){
		out[i]=alphabet[brute[i]];
	}
	out[wordLen]='\0';
}

int any(int *list, int listSize){
	for(int i=0;i<listSize;i++){
		if(list[i])return 1;
	}
	return 0;
}

void printWork(char *work, int workLen){
	char *out[] = { "false", "true" };
	for(int i=0;i<workLen;i++){
		printf("%s ", out[work[i]]);
	}
	printf("\n");
}

__global__ void searchPart(char *targetString, char *alphabet, char *brutePart, int workSize, int wordLen, int alphabetLen, int* results){
	// Go to the start of my works
	int workerId = threadIdx.x;
	// assume false, change if needed
	results[workerId] = 0;
	int incRes = 0;
	
	// printf("DEVICE BRUTE [");
	// for(int i=0;i<wordLen;i++)printf("%d, ", brutePart[i]);
	// printf("]\n");

	k_bruteIncrement(brutePart, alphabetLen, wordLen, workSize*workerId, &incRes);
	if(!incRes){
		return;
	}
	int count = 0;
	char* out = (char *) malloc((wordLen + 1)* sizeof(char));
	while(1) {
		if(count>=workSize) break;
		bruteToString(brutePart, wordLen, alphabet, out);
		int cmpRes = 1;
		cudaStrCmp(out, targetString, wordLen, &cmpRes);
		if(cmpRes == 1) { 
			results[workerId] = 1;
			break;
		}
		count +=1;
		incRes = 0;
		k_bruteIncrement(brutePart, alphabetLen, wordLen, 1, &incRes);
		if(!incRes) {
			break;
        }
	}
	free(out);
}
int search(char *targetString, char *alphabet, int numWorkers, int workSize){
	int wordLen = strlen(targetString);
    int alphabetLen = strlen(alphabet);
    int size = wordLen*sizeof(char);
    int alphabetSize = alphabetLen*sizeof(char);

    char *k_alphabet;
    int *k_alphabetLen;
    int *k_wordLen;
    char *k_targetString;

    hipMallocManaged(&k_alphabet, alphabetSize);
    hipMallocManaged(&k_alphabetLen, sizeof(int));
    hipMallocManaged(&k_wordLen, sizeof(int));
    hipMallocManaged(&k_targetString, size);
    hipMemcpy(k_alphabet, alphabet, alphabetSize, hipMemcpyDefault );
	hipMemcpy(k_targetString, targetString, size, hipMemcpyDefault );
	*k_alphabetLen = strlen(alphabet);
	*k_wordLen = strlen(targetString);

	// printf("HOST: Alphabet %s, Target %s\n", k_alphabet, k_targetString);

	char brute [wordLen];
	for(int i=0;i<wordLen;i++)brute[i]=0; // [0,0,0,...0]



	char* k_brutePart;
	hipMalloc(&k_brutePart, size);

	int* k_results;
	hipMallocManaged(&k_results, numWorkers* sizeof(int));

	int* results = (int*)malloc(sizeof(int) * numWorkers);
	while(1){
		
		// printf("HOST BRUTE [");
		// for(int i=0;i<wordLen;i++)printf("%d, ", brute[i]);
		// printf("]\n");

		hipMemcpy(k_brutePart, brute, size, hipMemcpyDefault );
		for(int i=0;i<numWorkers;i++) k_results[i] = 0;

        searchPart<<<1, numWorkers>>>(k_targetString, k_alphabet, k_brutePart, workSize, *k_wordLen, *k_alphabetLen, k_results);
		
		// printWork(work, numWorkers);

		// Wait for GPU to finish before accessing on host
		hipDeviceSynchronize();

		// printf("Results: ");
		// for (int i=0; i < numWorkers; i++) printf("%d, ", k_results[i]);
		// printf("\n");

		if(any(k_results, numWorkers)) return 1;

		// advance to the next major chunk of work
		// int bruteIncrement(char* brute, int alphabetLen, int wordLen, int incrementBy) {
		if(!bruteIncrement(brute, alphabetLen, wordLen, workSize*numWorkers)){
			break;
		}
	}
	hipFree(k_alphabet);
	hipFree(k_alphabetLen);
	hipFree(k_wordLen);
	hipFree(k_targetString);
	hipFree(k_brutePart);
	hipFree(k_results);
	return 0;
}

int main( int argc, char** argv) {
	char *targetString = argv[1];
	char *alphabet = argv[2];
	int numWorkers = atoi(argv[3]);
	int workSize = atoi(argv[4]);
	printf("Looking for %s in [%s]...\n", targetString, alphabet);
	if(search(targetString, alphabet, numWorkers, workSize)){
		printf("Found\n");
	} else {
		printf("Notfound\n");
	}

	return 0;
}