
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

const char TRUE = 1;
const char FALSE = 0;


int bruteIncrement(char* brute, int alphabetLen, int wordLen, int incrementBy) {
	int i = 0;
	while(incrementBy > 0 && i < wordLen) {
		int add = incrementBy + brute[i];
		brute[i] = (char)(add % alphabetLen);
		incrementBy = add / alphabetLen;
		i++;
	}
	return incrementBy == 0; 
}

__device__ void cudaStrCmp(char *a, char *b, int len, int* res) {
	int workerId = threadIdx.x;
	for (int i = 0; i < len; i++) {
		if (a[i] != b[i]) {
			*res = 0;
			return;
		}
	}
}

__device__ void k_bruteIncrement(char* brute, int alphabetLen, int wordLen, int incrementBy, int *incRes) {
	int i = 0;
	int workerId = threadIdx.x;
	
	while(incrementBy > 0 && i < wordLen) {
		int add = incrementBy + brute[i]; 
		brute[i] = (char)(add % alphabetLen); 
		incrementBy = add / alphabetLen;
		i++;
	}
	*incRes = incrementBy == 0;
}

__device__ void bruteToString(char *brute, int wordLen, char *alphabet, char *out){
	for(int i=0;i<wordLen;i++){
		out[i]=alphabet[brute[i]];
	}
	out[wordLen]='\0';
}

int any(int *list, int listSize){
	for(int i=0;i<listSize;i++){
		if(list[i])return TRUE;
	}
	return FALSE;
}

__global__ void searchPart(char *targetString, char *alphabet, char *brutePart, int workSize, int wordLen, int alphabetLen, int* results){
	int workerId = threadIdx.x;
	results[workerId] = 0;
	int incRes = FALSE;

	// Receive start of latest section (WORKER * WORKSIZE), create local copy
	char* t_brutePart = (char *) malloc((wordLen)* sizeof(char));
	for (int i = 0; i < wordLen; i++) t_brutePart[i] = brutePart[i];
	
	// Increment to start of this thread's chunk (WORKSIZE)
	k_bruteIncrement(t_brutePart, alphabetLen, wordLen, workSize*workerId, &incRes);

	if(!incRes){
		return;
	}
	int count = 0;
	char* out = (char *) malloc((wordLen + 1)* sizeof(char));

	// Increment by one and compare strs after every iteration
	while(1) {
		if(count>=workSize) {
			break;
		}
		bruteToString(t_brutePart, wordLen, alphabet, out);
		int cmpRes = 1;
		cudaStrCmp(out, targetString, wordLen, &cmpRes);
		if(cmpRes == 1) { 
			results[workerId] = 1;
			break;
		}
		count +=1;
		incRes = 0;
		k_bruteIncrement(t_brutePart, alphabetLen, wordLen, 1, &incRes);
		if(!incRes) {
			break;
        }
	}
	free(out);
	free(t_brutePart);
}
int search(char *targetString, char *alphabet, int numWorkers, int workSize){
	int wordLen = strlen(targetString);
    int alphabetLen = strlen(alphabet);
    int size = wordLen*sizeof(char);
    int alphabetSize = alphabetLen*sizeof(char);

    char *k_alphabet;
    int *k_alphabetLen;
    int *k_wordLen;
    char *k_targetString;

    hipMallocManaged(&k_alphabet, alphabetSize);
    hipMallocManaged(&k_alphabetLen, sizeof(int));
    hipMallocManaged(&k_wordLen, sizeof(int));
    hipMallocManaged(&k_targetString, size);
    hipMemcpy(k_alphabet, alphabet, alphabetSize, hipMemcpyDefault );
	hipMemcpy(k_targetString, targetString, size, hipMemcpyDefault );
	*k_alphabetLen = strlen(alphabet);
	*k_wordLen = strlen(targetString);

	char brute [wordLen];
	for(int i=0;i<wordLen;i++)brute[i]=0; 

	char* k_brutePart;
	hipMalloc(&k_brutePart, size);

	int* k_results;
	hipMallocManaged(&k_results, numWorkers* sizeof(int));

	int* results = (int*)malloc(sizeof(int) * numWorkers);
	// Every iteration, increment brute (WORKERS * WORKSIZE) times
	while(1){

		hipMemcpy(k_brutePart, brute, size, hipMemcpyDefault );
		for(int i=0;i<numWorkers;i++) k_results[i] = 0;

		// Divide the section into chunks to be worked on in parallel
        searchPart<<<1, numWorkers>>>(k_targetString, k_alphabet, k_brutePart, workSize, *k_wordLen, *k_alphabetLen, k_results);
		
		// Wait for GPU to finish before accessing on host
		hipDeviceSynchronize();

		if(any(k_results, numWorkers)) return 1;

		// advance to the next major chunk of work
		if(!bruteIncrement(brute, alphabetLen, wordLen, workSize*numWorkers)){
			break;
		}
	}
	hipFree(k_alphabet);
	hipFree(k_alphabetLen);
	hipFree(k_wordLen);
	hipFree(k_targetString);
	hipFree(k_brutePart);
	hipFree(k_results);
	return 0;
}

int main( int argc, char** argv) {
	char *targetString = argv[1];
	char *alphabet = argv[2];
	int numWorkers = atoi(argv[3]);
	int workSize = atoi(argv[4]);
	printf("Looking for %s in [%s]...\n", targetString, alphabet);
	if(search(targetString, alphabet, numWorkers, workSize)){
		printf("Found\n");
	} else {
		printf("Notfound\n");
	}

	return 0;
}